#include "hip/hip_runtime.h"
/*
NMS implementation in CUDA from pytorch framework
(https://github.com/pytorch/vision/tree/master/torchvision/csrc/cuda on Nov 13 2019)

Adapted for additional 3D capability by G. Ramien, DKFZ Heidelberg
*/

#include <torch/extension.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>

#include "cuda_helpers.h"

#include <iostream>
#include <vector>

int const threadsPerBlock = sizeof(unsigned long long) * 8;

template <typename T>
__device__ inline float devIoU(T const* const a, T const* const b) {
  // a, b hold box coords as (y1, x1, y2, x2) with y1 < y2 etc.
  T bottom = max(a[0], b[0]), top = min(a[2], b[2]);
  T left = max(a[1], b[1]), right = min(a[3], b[3]);
  T width = max(right - left, (T)0), height = max(top - bottom, (T)0);
  T interS = width * height;

  T Sa = (a[2] - a[0]) * (a[3] - a[1]);
  T Sb = (b[2] - b[0]) * (b[3] - b[1]);

  return interS / (Sa + Sb - interS);
}

template <typename T>
__device__ inline float devIoU_3d(T const* const a, T const* const b) {
  // a, b hold box coords as (y1, x1, y2, x2, z1, z2) with y1 < y2 etc.
  // get coordinates of intersection, calc intersection
  T bottom = max(a[0], b[0]), top = min(a[2], b[2]);
  T left = max(a[1], b[1]), right = min(a[3], b[3]);
  T front = max(a[4], b[4]), back = min(a[5], b[5]);
  T width = max(right - left, (T)0), height = max(top - bottom, (T)0);
  T depth = max(back - front, (T)0);
  T interS = width * height * depth;
  // calc separate boxes volumes
  T Sa = (a[2] - a[0]) * (a[3] - a[1]) * (a[5] - a[4]);
  T Sb = (b[2] - b[0]) * (b[3] - b[1]) * (b[5] - b[4]);

  return interS / (Sa + Sb - interS);
}


template <typename T>
__global__ void nms_kernel(const int n_boxes, const float iou_threshold, const T* dev_boxes,
    unsigned long long* dev_mask) {
  const int row_start = blockIdx.y;
  const int col_start = blockIdx.x;

  // if (row_start > col_start) return;
  const int row_size =
      min(n_boxes - row_start * threadsPerBlock, threadsPerBlock);
  const int col_size =
      min(n_boxes - col_start * threadsPerBlock, threadsPerBlock);

  __shared__ T block_boxes[threadsPerBlock * 4];
  if (threadIdx.x < col_size) {
    block_boxes[threadIdx.x * 4 + 0] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 4 + 0];
    block_boxes[threadIdx.x * 4 + 1] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 4 + 1];
    block_boxes[threadIdx.x * 4 + 2] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 4 + 2];
    block_boxes[threadIdx.x * 4 + 3] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 4 + 3];
  }
  __syncthreads();

  if (threadIdx.x < row_size) {
    const int cur_box_idx = threadsPerBlock * row_start + threadIdx.x;
    const T* cur_box = dev_boxes + cur_box_idx * 4;
    int i = 0;
    unsigned long long t = 0;
    int start = 0;
    if (row_start == col_start) {
      start = threadIdx.x + 1;
    }
    for (i = start; i < col_size; i++) {
      if (devIoU<T>(cur_box, block_boxes + i * 4) > iou_threshold) {
        t |= 1ULL << i;
      }
    }
    const int col_blocks = at::cuda::ATenCeilDiv(n_boxes, threadsPerBlock);
    dev_mask[cur_box_idx * col_blocks + col_start] = t;
  }
}


template <typename T>
__global__ void nms_kernel_3d(const int n_boxes, const float iou_threshold, const T* dev_boxes,
    unsigned long long* dev_mask) {
  const int row_start = blockIdx.y;
  const int col_start = blockIdx.x;

  // if (row_start > col_start) return;
  const int row_size =
      min(n_boxes - row_start * threadsPerBlock, threadsPerBlock);
  const int col_size =
      min(n_boxes - col_start * threadsPerBlock, threadsPerBlock);

  __shared__ T block_boxes[threadsPerBlock * 6];
  if (threadIdx.x < col_size) {
    block_boxes[threadIdx.x * 6 + 0] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 6 + 0];
    block_boxes[threadIdx.x * 6 + 1] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 6 + 1];
    block_boxes[threadIdx.x * 6 + 2] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 6 + 2];
    block_boxes[threadIdx.x * 6 + 3] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 6 + 3];
    block_boxes[threadIdx.x * 6 + 4] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 6 + 4];
    block_boxes[threadIdx.x * 6 + 5] =
        dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 6 + 5];
  }
  __syncthreads();

  if (threadIdx.x < row_size) {
    const int cur_box_idx = threadsPerBlock * row_start + threadIdx.x;
    const T* cur_box = dev_boxes + cur_box_idx * 6;
    int i = 0;
    unsigned long long t = 0;
    int start = 0;
    if (row_start == col_start) {
      start = threadIdx.x + 1;
    }
    for (i = start; i < col_size; i++) {
      if (devIoU_3d<T>(cur_box, block_boxes + i * 6) > iou_threshold) {
        t |= 1ULL << i;
      }
    }
    const int col_blocks = at::cuda::ATenCeilDiv(n_boxes, threadsPerBlock);
    dev_mask[cur_box_idx * col_blocks + col_start] = t;
  }
}


at::Tensor nms_cuda(const at::Tensor& dets, const at::Tensor& scores, float iou_threshold) {
  /* dets expected as (n_dets, dim) where dim=4 in 2D, dim=6 in 3D */
  AT_ASSERTM(dets.type().is_cuda(), "dets must be a CUDA tensor");
  AT_ASSERTM(scores.type().is_cuda(), "scores must be a CUDA tensor");
  at::cuda::CUDAGuard device_guard(dets.device());

  bool is_3d = dets.size(1) == 6;
  auto order_t = std::get<1>(scores.sort(0, /* descending=*/true));
  auto dets_sorted = dets.index_select(0, order_t);

  int dets_num = dets.size(0);

  const int col_blocks = at::cuda::ATenCeilDiv(dets_num, threadsPerBlock);

  at::Tensor mask =
      at::empty({dets_num * col_blocks}, dets.options().dtype(at::kLong));

  dim3 blocks(col_blocks, col_blocks);
  dim3 threads(threadsPerBlock);
  hipStream_t stream = at::cuda::getCurrentCUDAStream();


  if (is_3d) {
  //std::cout << "performing NMS on 3D boxes in CUDA" << std::endl;
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      dets_sorted.type(), "nms_kernel_cuda", [&] {
        nms_kernel_3d<scalar_t><<<blocks, threads, 0, stream>>>(
            dets_num,
            iou_threshold,
            dets_sorted.data_ptr<scalar_t>(),
            (unsigned long long*)mask.data_ptr<int64_t>());
      });
   }
   else {
   AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      dets_sorted.type(), "nms_kernel_cuda", [&] {
        nms_kernel<scalar_t><<<blocks, threads, 0, stream>>>(
            dets_num,
            iou_threshold,
            dets_sorted.data_ptr<scalar_t>(),
            (unsigned long long*)mask.data_ptr<int64_t>());
      });

   }

  at::Tensor mask_cpu = mask.to(at::kCPU);
  unsigned long long* mask_host = (unsigned long long*)mask_cpu.data_ptr<int64_t>();

  std::vector<unsigned long long> remv(col_blocks);
  memset(&remv[0], 0, sizeof(unsigned long long) * col_blocks);

  at::Tensor keep =
      at::empty({dets_num}, dets.options().dtype(at::kLong).device(at::kCPU));
  int64_t* keep_out = keep.data_ptr<int64_t>();

  int num_to_keep = 0;
  for (int i = 0; i < dets_num; i++) {
    int nblock = i / threadsPerBlock;
    int inblock = i % threadsPerBlock;

    if (!(remv[nblock] & (1ULL << inblock))) {
      keep_out[num_to_keep++] = i;
      unsigned long long* p = mask_host + i * col_blocks;
      for (int j = nblock; j < col_blocks; j++) {
        remv[j] |= p[j];
      }
    }
  }

  AT_CUDA_CHECK(hipGetLastError());
  return order_t.index(
      {keep.narrow(/*dim=*/0, /*start=*/0, /*length=*/num_to_keep)
           .to(order_t.device(), keep.scalar_type())});
}